#include "hip/hip_runtime.h"
#include "caffe/layers/reverse_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void reverse_gpu(const int nthreads, const Dtype* from_data, Dtype* to_data, 
	const int* counts, const int axis_count, const int axis) {
  CUDA_KERNEL_LOOP(index, nthreads) {
  	int ind=(index/counts[axis])%axis_count;
  	int to_index=counts[axis]*(axis_count-2*ind-1)+index;
  	*(to_data+to_index)=*(from_data+index);
  }
}

template <typename Ftype, typename Btype>
void ReverseLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom, 
		const vector<Blob*>& top) {
	const int nthreads=bottom[0]->count();
	reverse_gpu<Ftype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom[0]->gpu_data<Ftype>(), top[0]->mutable_gpu_data<Ftype>(), 
        bottom_counts_.gpu_data(), bottom[0]->shape(axis_), axis_);
}



INSTANTIATE_LAYER_GPU_FORWARD_ONLY_FB(ReverseLayer);

}  // namespace caffe
